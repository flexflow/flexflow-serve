/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/kernels/gemm_impl.h"
#include "flexflow/utils/cuda_helper.h"
#include <memory>

namespace Internal {

GemmEngine::GemmEngine(hipblasHandle_t blas_,
                       hipblasLtHandle_t blasLt_,
                       hipDeviceProp_t *device_prop_,
                       size_t workspace_size_) {
  blas = blas_;
  blasLt = blasLt_;
  if (device_prop_ == nullptr) {
    device_prop = new hipDeviceProp_t;
  } else {
    device_prop = device_prop_;
  }
  workspace_size = workspace_size_;
  workspace = nullptr;
}

void GemmEngine::assign_workspace(void *workspace_, size_t workspace_size_) {
  assert(workspace_size_ >= workspace_size);
  workspace = workspace_;
}

template <typename Dtype>
void GemmEngine::gemm_internal(hipblasOperation_t transa,
                               hipblasOperation_t transb,
                               int64_t m,
                               int64_t n,
                               int64_t k,
                               Dtype alpha,
                               Dtype const *a,
                               int64_t lda,
                               Dtype const *b,
                               int64_t ldb,
                               Dtype beta,
                               Dtype *c,
                               int64_t ldc,
                               hipStream_t stream) {
  static_assert(false && sizeof(Dtype), "gemm_internal: not implemented");
}

#ifdef USE_CUBLASLT
/* Implementations for gemm_internal_cublaslt */
template <typename T, hipblasStatus_t (*destructor)(T *)>
struct CuBlasLtDeleter {
  void operator()(T *x) {
    if (x != nullptr) {
      checkCUDA(destructor(x));
    }
  }
};

template <typename T, hipblasStatus_t (*destructor)(T *)>
class CuBlasLtDescriptor {
public:
  T *descriptor() const {
    return descriptor_.get();
  }
  T *descriptor() {
    return descriptor_.get();
  }

protected:
  std::unique_ptr<T, CuBlasLtDeleter<T, destructor>> descriptor_;
};

class CuBlasLtMatmulDescriptor
    : public CuBlasLtDescriptor<hipblasLtMatmulDescOpaque_t,
                                &hipblasLtMatmulDescDestroy> {
public:
  CuBlasLtMatmulDescriptor(hipblasComputeType_t compute_type,
                           hipDataType scale_type) {
    hipblasLtMatmulDesc_t raw_descriptor = nullptr;
    checkCUDA(
        hipblasLtMatmulDescCreate(&raw_descriptor, compute_type, scale_type));
    descriptor_.reset(raw_descriptor);
  }
  template <typename T>
  inline void setAttribute(hipblasLtMatmulDescAttributes_t attr, const T value) {
    // NOLINTNEXTLINE(bugprone-sizeof-expression)
    checkCUDA(::hipblasLtMatmulDescSetAttribute(
        descriptor(), attr, &value, sizeof(T)));
  }
};

class CuBlasLtMatrixLayout
    : public CuBlasLtDescriptor<hipblasLtMatrixLayoutOpaque_t,
                                &hipblasLtMatrixLayoutDestroy> {
public:
  CuBlasLtMatrixLayout(hipDataType type,
                       uint64_t rows,
                       uint64_t cols,
                       int64_t ld,
                       bool t = false) {
    hipblasLtMatrixLayout_t raw_descriptor = nullptr;
    checkCUDA(hipblasLtMatrixLayoutCreate(
        &raw_descriptor, type, t ? cols : rows, t ? rows : cols, ld));
    descriptor_.reset(raw_descriptor);
  }
  template <typename T>
  inline void setAttribute(hipblasLtMatrixLayoutAttribute_t attr,
                           const T value) {
    checkCUDA(::hipblasLtMatrixLayoutSetAttribute(
        descriptor(), attr, &value, sizeof(T)));
  }
};

class CuBlasLtMatmulPreference
    : public CuBlasLtDescriptor<hipblasLtMatmulPreferenceOpaque_t,
                                &hipblasLtMatmulPreferenceDestroy> {
public:
  CuBlasLtMatmulPreference() {
    hipblasLtMatmulPreference_t raw_descriptor = nullptr;
    checkCUDA(hipblasLtMatmulPreferenceCreate(&raw_descriptor));
    descriptor_.reset(raw_descriptor);
  }
  template <typename T>
  inline void setAttribute(hipblasLtMatmulPreferenceAttributes_t attr,
                           const T value) {
    checkCUDA(::hipblasLtMatmulPreferenceSetAttribute(
        descriptor(), attr, &value, sizeof(T)));
  }
};

inline uint32_t _getAlignment(uintptr_t address) {
  // alignment are in bytes
  uint32_t alignment = 256;
  for (;; alignment /= 2) {
    if (!(address % alignment)) {
      return alignment;
    }
  }
}

template <typename Dtype>
inline void gemm_internal_cublaslt(hipblasLtHandle_t handle,
                                   hipDeviceProp_t *prop,
                                   void *workspace,
                                   size_t workspace_size,
                                   hipblasOperation_t transa,
                                   hipblasOperation_t transb,
                                   int64_t m,
                                   int64_t n,
                                   int64_t k,
                                   Dtype alpha,
                                   Dtype const *a,
                                   int64_t lda,
                                   Dtype const *b,
                                   int64_t ldb,
                                   Dtype beta,
                                   Dtype *c,
                                   int64_t ldc,
                                   hipStream_t stream) {
  assert(workspace != nullptr && "workspace must be provided.");
  hipDataType abcType = HIP_R_32F;
  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F;
  hipDataType scaleType = HIP_R_32F;
  if constexpr (std::is_same_v<Dtype, double>) {
    abcType = HIP_R_64F;
    computeType = HIPBLAS_COMPUTE_64F;
    scaleType = HIP_R_64F;
  } else if constexpr (std::is_same_v<Dtype, float>) {
    computeType = HIPBLAS_COMPUTE_32F_FAST_TF32;
  } else if constexpr (std::is_same_v<Dtype, half>) {
    abcType = HIP_R_16F;
    computeType = HIPBLAS_COMPUTE_16F;
  } else {
    static_assert(false && sizeof(Dtype),
                  "bgemm_internal_cublaslt: not implemented");
  }

  CuBlasLtMatmulDescriptor computeDesc(computeType, scaleType);
  computeDesc.setAttribute(HIPBLASLT_MATMUL_DESC_TRANSA, transa);
  computeDesc.setAttribute(HIPBLASLT_MATMUL_DESC_TRANSB, transb);
  CuBlasLtMatrixLayout Adesc(abcType, m, k, lda, transa == HIPBLAS_OP_T);
  CuBlasLtMatrixLayout Bdesc(abcType, k, n, ldb, transb == HIPBLAS_OP_T);
  CuBlasLtMatrixLayout Cdesc(abcType, m, n, ldc);

  CuBlasLtMatmulPreference preference;
  preference.setAttribute(HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                          workspace_size);

  uint32_t a_alignment = _getAlignment(reinterpret_cast<uintptr_t>(a));
  uint32_t b_alignment = _getAlignment(reinterpret_cast<uintptr_t>(b));
  uint32_t c_alignment = _getAlignment(reinterpret_cast<uintptr_t>(c));
  preference.setAttribute(CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_A_BYTES,
                          a_alignment);
  preference.setAttribute(CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_B_BYTES,
                          b_alignment);
  preference.setAttribute(CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_C_BYTES,
                          c_alignment);

  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  int returnedResult = 0;
  checkCUDA(hipblasLtMatmulAlgoGetHeuristic(handle,
                                           computeDesc.descriptor(),
                                           Adesc.descriptor(),
                                           Bdesc.descriptor(),
                                           Cdesc.descriptor(),
                                           Cdesc.descriptor(),
                                           preference.descriptor(),
                                           1,
                                           &heuristicResult,
                                           &returnedResult));
  if (returnedResult == 0) {
    assert(false && "cuBLASLt failed to find a valid algorithm.");
  }

  checkCUDA(hipblasLtMatmul(handle,
                           computeDesc.descriptor(),
                           &alpha,
                           a,
                           Adesc.descriptor(),
                           b,
                           Bdesc.descriptor(),
                           &beta,
                           c,
                           Cdesc.descriptor(),
                           c,
                           Cdesc.descriptor(),
                           &heuristicResult.algo,
                           workspace,
                           workspace_size,
                           stream));
}
#else
/* Implementations for gemm_internal_cublas */
template <typename Dtype>
inline void gemm_internal_cublas(hipblasHandle_t handle,
                                 hipDeviceProp_t *prop,
                                 hipblasOperation_t transa,
                                 hipblasOperation_t transb,
                                 int64_t m,
                                 int64_t n,
                                 int64_t k,
                                 Dtype alpha,
                                 Dtype const *a,
                                 int64_t lda,
                                 Dtype const *b,
                                 int64_t ldb,
                                 Dtype beta,
                                 Dtype *c,
                                 int64_t ldc,
                                 hipStream_t stream) {
  static_assert(false && sizeof(Dtype),
                "gemm_internal_cublas: not implemented");
}

template <>
void gemm_internal_cublas<double>(hipblasHandle_t handle,
                                  hipDeviceProp_t *prop,
                                  hipblasOperation_t transa,
                                  hipblasOperation_t transb,
                                  int64_t m,
                                  int64_t n,
                                  int64_t k,
                                  double alpha,
                                  double const *a,
                                  int64_t lda,
                                  double const *b,
                                  int64_t ldb,
                                  double beta,
                                  double *c,
                                  int64_t ldc,
                                  hipStream_t stream) {
  checkCUDA(hipblasDgemm(
      handle, transa, transb, m, n, k, &alpha, a, lda, b, ldb, &beta, c, ldc));
}

template <>
void gemm_internal_cublas<float>(hipblasHandle_t handle,
                                 hipDeviceProp_t *prop,
                                 hipblasOperation_t transa,
                                 hipblasOperation_t transb,
                                 int64_t m,
                                 int64_t n,
                                 int64_t k,
                                 float alpha,
                                 float const *a,
                                 int64_t lda,
                                 float const *b,
                                 int64_t ldb,
                                 float beta,
                                 float *c,
                                 int64_t ldc,
                                 hipStream_t stream) {
  checkCUDA(hipblasSgemm(
      handle, transa, transb, m, n, k, &alpha, a, lda, b, ldb, &beta, c, ldc));
  // checkCUDA(hipblasGemmEx(
  //     handle,
  //     transa,
  //     transb,
  //     m,
  //     n,
  //     k,
  //     &alpha,
  //     a,
  //     HIP_R_32F,
  //     lda,
  //     b,
  //     HIP_R_32F,
  //     ldb,
  //     &beta,
  //     c,
  //     HIP_R_32F,
  //     ldc,
  //     HIPBLAS_COMPUTE_32F_FAST_16F,
  //     CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

template <>
void gemm_internal_cublas<half>(hipblasHandle_t handle,
                                hipDeviceProp_t *prop,
                                hipblasOperation_t transa,
                                hipblasOperation_t transb,
                                int64_t m,
                                int64_t n,
                                int64_t k,
                                half alpha,
                                half const *a,
                                int64_t lda,
                                half const *b,
                                int64_t ldb,
                                half beta,
                                half *c,
                                int64_t ldc,
                                hipStream_t stream) {
  if (prop->major >= 5) {
    // Disallow fp16 reductions that could lead to unexpected overflow issues.
    // hipblasMath_t cublas_flags = HIPBLAS_DEFAULT_MATH;
    // if (!at::globalContext().allowFP16ReductionCuBLAS()) {
    //   cublas_flags = static_cast<hipblasMath_t>(cublas_flags |
    //   HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION);
    // }
    // checkCUDA(hipblasSetMathMode(handle, cublas_flags));
    checkCUDA(hipblasGemmEx(handle,
                           transa,
                           transb,
                           m,
                           n,
                           k,
                           &alpha,
                           a,
                           HIP_R_16F,
                           lda,
                           b,
                           HIP_R_16F,
                           ldb,
                           &beta,
                           c,
                           HIP_R_16F,
                           ldc,
                           HIPBLAS_COMPUTE_16F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // checkCUDA(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
  } else {
    float falpha = alpha;
    float fbeta = beta;
    checkCUDA(cublasSgemmEx(handle,
                            transa,
                            transb,
                            m,
                            n,
                            k,
                            &falpha,
                            a,
                            HIP_R_16F,
                            lda,
                            b,
                            HIP_R_16F,
                            ldb,
                            &fbeta,
                            c,
                            HIP_R_16F,
                            ldc));
  }
}
#endif

template <>
void GemmEngine::gemm_internal(hipblasOperation_t transa,
                               hipblasOperation_t transb,
                               int64_t m,
                               int64_t n,
                               int64_t k,
                               double alpha,
                               double const *a,
                               int64_t lda,
                               double const *b,
                               int64_t ldb,
                               double beta,
                               double *c,
                               int64_t ldc,
                               hipStream_t stream) {
#ifdef USE_CUBLASLT
  gemm_internal_cublaslt(blasLt,
                         device_prop,
                         workspace,
                         workspace_size,
                         transa,
                         transb,
                         m,
                         n,
                         k,
                         alpha,
                         a,
                         lda,
                         b,
                         ldb,
                         beta,
                         c,
                         ldc,
                         stream);
#else
  gemm_internal_cublas(blas,
                       device_prop,
                       transa,
                       transb,
                       m,
                       n,
                       k,
                       alpha,
                       a,
                       lda,
                       b,
                       ldb,
                       beta,
                       c,
                       ldc,
                       stream);
#endif
}

template <>
void GemmEngine::gemm_internal(hipblasOperation_t transa,
                               hipblasOperation_t transb,
                               int64_t m,
                               int64_t n,
                               int64_t k,
                               float alpha,
                               float const *a,
                               int64_t lda,
                               float const *b,
                               int64_t ldb,
                               float beta,
                               float *c,
                               int64_t ldc,
                               hipStream_t stream) {
#ifdef USE_CUBLASLT
  gemm_internal_cublaslt(blasLt,
                         device_prop,
                         workspace,
                         workspace_size,
                         transa,
                         transb,
                         m,
                         n,
                         k,
                         alpha,
                         a,
                         lda,
                         b,
                         ldb,
                         beta,
                         c,
                         ldc,
                         stream);
#else
  gemm_internal_cublas(blas,
                       device_prop,
                       transa,
                       transb,
                       m,
                       n,
                       k,
                       alpha,
                       a,
                       lda,
                       b,
                       ldb,
                       beta,
                       c,
                       ldc,
                       stream);
#endif
}

template <>
void GemmEngine::gemm_internal(hipblasOperation_t transa,
                               hipblasOperation_t transb,
                               int64_t m,
                               int64_t n,
                               int64_t k,
                               half alpha,
                               half const *a,
                               int64_t lda,
                               half const *b,
                               int64_t ldb,
                               half beta,
                               half *c,
                               int64_t ldc,
                               hipStream_t stream) {
#ifdef USE_CUBLASLT
  gemm_internal_cublaslt(blasLt,
                         device_prop,
                         workspace,
                         workspace_size,
                         transa,
                         transb,
                         m,
                         n,
                         k,
                         alpha,
                         a,
                         lda,
                         b,
                         ldb,
                         beta,
                         c,
                         ldc,
                         stream);
#else
  gemm_internal_cublas(blas,
                       device_prop,
                       transa,
                       transb,
                       m,
                       n,
                       k,
                       alpha,
                       a,
                       lda,
                       b,
                       ldb,
                       beta,
                       c,
                       ldc,
                       stream);
#endif
}
} // namespace Internal
